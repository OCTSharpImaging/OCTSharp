#include "hip/hip_runtime.h"
#define _SIZE_T_DEFOMED
#ifndef __HIPCC__
#define __HIPCC__
#endif
#ifndef __cplusplus
#define __cplusplus
#endif
#define CUDA_CODE_CU

//#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <>
#include <builtin_types.h>
#include <vector_functions.h>
#include "float.h"
#include <hipfft/hipfft.h>
#include <hip/hip_fp16.h>

extern "C" 
{
	__global__ void inputCastKernel(hipfftComplex* output, const void* input, const int inputBitdepth, int samplePerBuffer)
	{
		int idx = threadIdx.x + blockIdx.x * blockDim.x;
			if (inputBitdepth <= 8) {
				unsigned char* in = (unsigned char*)input;
				output[idx].x = __uint2float_rd(in[idx]);
			}
			else if (inputBitdepth > 8 && inputBitdepth <= 16) {
				unsigned short* in = (unsigned short*)input;
				output[idx].x = __uint2float_rd(in[idx]);
			}
			else {
				unsigned int* in = (unsigned int*)input;
				output[idx].x = __uint2float_rd(in[idx]);
			}	
		output[idx].y = 0;
	}
}	

extern "C"
{
	__global__ void plotSpectrumKernel(unsigned char* output, unsigned short* in, int width, int height)
	{
		int idx = threadIdx.x + blockIdx.x * blockDim.x;
		output[idx] = 0;

		int pixelIdx = idx % width;
		int pixelOffset = in[pixelIdx];	
		
		int spectrumIndex = idx % height;
		output[spectrumIndex * height + pixelOffset] = 255;
	}
}

extern "C"
{
	__global__ void spectrumByte2FloatKernel(float* output, unsigned char* input)
	{
		int idx = threadIdx.x + blockIdx.x * blockDim.x;
		output[idx] = __uint2float_rd(input[idx]);
	}
}

extern "C"
{
	__global__ void spectrumFloat2ByteKernel(unsigned char* output, float* input)
	{
		int idx = threadIdx.x + blockIdx.x * blockDim.x;
		output[idx] = __float2uint_rd(input[idx]);
	}
}

// remove DC by kernel
//extern "C" 
//{
//	__global__ void meanALineSubstractionKernel(hipfftComplex *out, float *in, int width, int samples) {
//		int idx = threadIdx.x + blockIdx.x * blockDim.x;
//		out[idx].x = out[idx].x - in[idx % width];
//	}
//}

// remove DC by cublast
extern "C" {
	__global__ void meanALineSubstractionKernel(hipfftComplex *out, hipfftComplex *in, int width, int height) {
		int idx = threadIdx.x + blockIdx.x * blockDim.x;
		out[idx].x = out[idx].x - in[idx % width].x/width;
	}
}

extern "C" 
{
	__global__ void CubicInterpretationKernal(hipfftComplex* out, hipfftComplex* in, const float*resampleCurve, const int width) {
		int idx = threadIdx.x + blockIdx.x * blockDim.x;
		int j = idx % width;
		int offset = idx - j;

		float x = resampleCurve[j];
		int x0 = (int)x;
		int x1 = x0 + 1;
		int x2 = x0 + 2;
		int x3 = x0 + 3;

		float f_x0 = in[offset + x0].x;
		float f_x1 = in[offset + x1].x;
		float f_x2 = in[offset + x2].x;
		float f_x3 = in[offset + x2].x;
		float b0 = f_x0;
		float b1 = f_x1 - f_x0;
		float b2 = ((f_x2 - f_x1) - b1) / (x2 - x0);
		float b3 = ((f_x3 - f_x2) - b2) / (x3 - x0);

		out[idx].x = b0 + b1 * (x - x0) + b2 * (x - x0)*(x - x1) + b3 * (x - x0)*(x - x1)*(x - x2);
		out[idx].y = 0;
	}
}

extern "C" {
	__global__ void modulusKernel(float *output, const hipfftComplex *input, const int nx, const int samples, const float max, const float min, const float coeff, const float addend) {
		int idx = threadIdx.x + blockIdx.x * blockDim.x;
		if (idx < samples / 2) {
			int lineIndex = idx / nx;
			int inputArrayIndex = lineIndex * nx + idx;	
			output[idx] = (sqrt((input[inputArrayIndex].x * input[inputArrayIndex].x) + (input[inputArrayIndex].y * input[inputArrayIndex].y)));
		}
	}
}

extern "C" 
{
	__global__ void dfsModulusKernel(float *output, hipfftComplex *input) 
	{
		int idx = threadIdx.x + blockIdx.x * blockDim.x;	
		output[idx] = 10.0f *log10f(sqrt((input[idx].x * input[idx].x) + (input[idx].y * input[idx].y)));
		//output[idx] = sqrt((input[idx].x * input[idx].x) + (input[idx].y * input[idx].y));
	}
}

extern "C" 
{
	__global__ void SumKernel(float *output, float *input)
	{
		int idx = threadIdx.x + blockIdx.x * blockDim.x;
		output[idx] = output[idx] + input[idx];
	}
}

extern "C"
{
	__global__ void AvgKernel(float *output, float *input, const int AvgNum)
	{
		int idx = threadIdx.x + blockIdx.x * blockDim.x;
		output[idx] = input[idx]/AvgNum;
	}
}

extern "C"
{
	__global__ void VariantKernel(float *output, float *frame1, float *frame2, float *sumframe, const int AvgNum)
	{
		int idx = threadIdx.x + blockIdx.x * blockDim.x;
		//output[idx] = (frame1[idx] - sumframe[idx]) * (frame1[idx] - sumframe[idx]) + (frame2[idx] - sumframe[idx]) * (frame2[idx] - sumframe[idx]);
		//output[idx] = (frame1[idx] - frame2[idx]) * (frame1[idx] - frame2[idx]);
		//output[idx] = output[idx]/AvgNum;

		//float avg = (frame1[idx] + frame2[idx]) / 2;
		//output[idx] = (frame1[idx] - avg) * (frame1[idx] - avg) + (frame2[idx] - avg) * (frame2[idx] - avg);
		
		output[idx] = abs(frame1[idx] - frame2[idx]);
	}
}

extern "C"
{
	__global__ void BScanOutputCastKernel(unsigned char* output, float* input, const float max, const float min, const float coeff)
	{
		int index = threadIdx.x + blockIdx.x * blockDim.x;
		float* in = input;

		output[index] = 10.0f * log10f(in[index]);
		if ((output[index] - min) / (max - min) >= 0.255) {
			output[index] = 255;//highest 8bit grayscale 			
		}
		else if ((output[index] - min) / (max - min) < 0) {
			output[index] = 0;//lowest 8bit grayscale
		}
		else {
			output[index] = coeff * (output[index] - min) / (max - min);
		}
	}
}

extern "C"
{
	__global__ void EnfaceOutputCastKernel(unsigned char *output, float *input, float max, float min, const float coeff)
	{
		int idx = threadIdx.x + blockIdx.x * blockDim.x;
		float* in = input;
		max = 150;
		min = 50;
		//output[idx] = input[idx];
		output[idx] = 10.0f * log10f(in[idx]);
		if ((output[idx] - min) / (max - min) >= 0.255) {
			output[idx] = 255;//highest 8bit grayscale 			
		}
		else if ((output[idx] - min) / (max - min) < 0) {
			output[idx] = 0;//lowest 8bit grayscale
		}
		else {
			output[idx] = coeff *(output[idx] - min) / (max - min);
		}
	}
}

extern "C"
{
	__global__ void resetKernel(float *output)
	{
		int idx = threadIdx.x + blockIdx.x * blockDim.x;
		output[idx] = 0;
	}
}

extern "C"
{
	__global__ void copyKernel(float *output, float *input)
	{
		int idx = threadIdx.x + blockIdx.x * blockDim.x;
		output[idx] = input[idx];
	}
}

extern "C"
{
	__global__ void AssignComplexOneKernel(hipfftComplex *out)
	{
		int idx = threadIdx.x + blockIdx.x * blockDim.x;
		out[idx].x = 1.0f;
		out[idx].y = 0.0f;
	}
}

//dipreciated functions
extern "C"
{
	__global__ void AverageFrameKernal(float *out, hipfftComplex *in, int width, int height, int samples)
	{
		int idx = threadIdx.x + blockIdx.x * blockDim.x;
		for (int i = 0; i < height; i++) {
			out[idx] += in[idx + i * width].x;
		}
		out[idx] /= height;
	}
}